#include <stdio.h>

#include "hip/hip_runtime.h"

__global__ void HelloFromGPU(void)
{
	printf("Hello World From GPU!\n");
}


extern "C" int Hello()
{
	HelloFromGPU <<<2,2>>> ();

	return 0;
}